#include "hip/hip_runtime.h"
//   Copyright 2017, Felice Pantaleo, CERN
//
//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at
//
//       http://www.apache.org/licenses/LICENSE-2.0
//
//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.



#include <chrono>
#include <thread>
#include <vector>
#include <iostream>
#include <cassert>
#include <tbb/concurrent_queue.h>
#include <omp.h>
#include <tuple>
#include <mutex>

#include "Event.h"
#include "GPUHitsAndDoublets.h"
#include "GPUCACell.h"
#include "parser.h"
#include "hip/hip_runtime.h"
#include "kernels.h"
#include "host_kernels.h"
static void show_usage(std::string name)
{
    std::cerr << "\nUsage: " << name << " <option(s)>" << " Options:\n"
            << "\t-h,--help\t\tShow this help message\n"
            << "\t-n <number of events>\tSpecify the number of events to process [default: as many there are in the file]\n"
            << "\t-i <inputFile>\tSpecify the path of the input ASCII file containing the events to process [default: ../input/parsed.out]\n"
            << std::endl;

}

int main(int argc, char** argv)
{

    if (argc < 2)
    {
        show_usage(argv[0]);
        return 1;
    }

    unsigned int maxEvents = 10;
    std::string inputFile = "../input/parsed_noPU_fix.txt";
    unsigned int numberOfCUDAStreams = 5;
    unsigned int numberOfEventsPerStreamPerIteration = 1;
    unsigned int numberOfIterations = 1;
    unsigned int numberOfCPUThreads = 1;

    for (int i = 1; i < argc; ++i)
    {
        std::string arg = argv[i];
        if ((arg == "-h") || (arg == "--help"))
        {
            show_usage(argv[0]);
            return 0;
        }
        else if (arg == "-n")
        {
            if (i + 1 < argc) // Make sure we aren't at the end of argv!
            {
                i++;
                std::istringstream ss(argv[i]);
                if (!(ss >> maxEvents))
                {
                    std::cerr << "Invalid number " << argv[i] << '\n';
                    exit(1);

                }
            }
        }
        else if (arg == "-i")
        {
            if (i + 1 < argc) // Make sure we aren't at the end of argv!
            {
                i++;
                std::istringstream ss(argv[i]);
                if (!(ss >> inputFile))
                {
                    std::cerr << "Invalid string " << argv[i] << '\n';
                    exit(1);

                }
            }
        }

        else if (arg == "-s")
        {
            if (i + 1 < argc) // Make sure we aren't at the end of argv!
            {
                i++;
                std::istringstream ss(argv[i]);
                if (!(ss >> numberOfCUDAStreams))
                {
                    std::cerr << "Invalid number " << argv[i] << '\n';
                    exit(1);

                }
            }
        }

        else if (arg == "-t")
        {
            if (i + 1 < argc) // Make sure we aren't at the end of argv!
            {
                i++;
                std::istringstream ss(argv[i]);
                if (!(ss >> numberOfIterations))
                {
                    std::cerr << "Invalid number " << argv[i] << '\n';
                    exit(1);

                }
            }
        }

        else if (arg == "-j")
        {
            if (i + 1 < argc) // Make sure we aren't at the end of argv!
            {
                i++;
                std::istringstream ss(argv[i]);
                if (!(ss >> numberOfCPUThreads))
                {
                    std::cerr << "Invalid number " << argv[i] << '\n';
                    exit(1);

                }
            }
        }

        else if (arg == "-b")
        {
            if (i + 1 < argc) // Make sure we aren't at the end of argv!
            {
                i++;
                std::istringstream ss(argv[i]);
                if (!(ss >> numberOfEventsPerStreamPerIteration))
                {
                    std::cerr << "Invalid number " << argv[i] << '\n';
                    exit(1);

                }
            }
        }

    }

    std::vector<HostEvent> hostEvents;
    Region* h_regionParams;
    hipHostMalloc(&h_regionParams, sizeof(Region));

    if (maxEvents > 0)
    {
        hostEvents.reserve(maxEvents);
    }

    constexpr unsigned int maxNumberOfQuadruplets = 3000;
    constexpr unsigned int maxCellsPerHit = 100;
    unsigned int maxNumberOfHits = 0;
    unsigned int maxNumberOfDoublets = 0;
    parseinputFile(inputFile, hostEvents, *h_regionParams, maxEvents, maxNumberOfHits,
            maxNumberOfDoublets);
    unsigned int nEvents = hostEvents.size();
    std::cout << "Correctly parsed file containing " << nEvents << " events." << std::endl;

//    std::cout << "max number of hits and doublets in file " << maxNumberOfHits << " "
//            << maxNumberOfDoublets << std::endl;
//
//    std::cout << "hostEvents contains " << hostEvents.size() << " events" <<  std::endl;
//
//    for(auto& ev: hostEvents)
//    {
//        std::cout << "event id " <<ev.eventId << " rootlayers " << ev.rootLayers.size() << " hitsLayers " << ev.hitsLayers.size() <<
//                " layerPairs " <<  ev.doublets.size() << std::endl;
//        for(auto& rl : ev.rootLayers)
//            std::cout<< "root layer: " << rl << std::endl;
//        for(auto& hl : ev.hitsLayers)
//        {
//            std::cout<< "hits on layer: " << hl.size << " " << hl.x.size() << std::endl;
//
//            for(int jj = 0; jj < hl.x.size(); ++jj)
//                std::cout<< "hit " << jj << hl.x[jj] << " " << hl.y[jj] << " " << hl.z[jj] << std::endl;
//
//
//        }
//        for(auto& dl : ev.doublets)
//            {
//                std::cout<< "doublets on layer pair: " << dl.innerLayerId << " " << dl.outerLayerId << " " << dl.size << " " << dl.indices.size() << std::endl;
//                    for(unsigned int i = 0; i<dl.size; ++i )
//                    {
//                        std::cout<< "\t hits in doublet: "<< i << " " <<  dl.indices[2*i] << " "<< dl.indices[2*i+1] <<" " << ev.hitsLayers[dl.innerLayerId].x[dl.indices[2*i]] << " "<< ev.hitsLayers[dl.innerLayerId].y[dl.indices[2*i]] << " "<<ev.hitsLayers[dl.innerLayerId].z[dl.indices[2*i]]<< " "
//                         << " \t\t" << ev.hitsLayers[dl.outerLayerId].x[dl.indices[2*i+1]] << " "<< ev.hitsLayers[dl.outerLayerId].y[dl.indices[2*i+1]] << " "<<ev.hitsLayers[dl.outerLayerId].z[dl.indices[2*i+1]]<< std::endl;
//                    }
//            }
//    }

    std::cout << "preallocating pinned memory on host" << std::endl;

    unsigned int eventsPreallocatedOnGPU = numberOfEventsPerStreamPerIteration
            * numberOfCUDAStreams;
    unsigned int maxNumberOfLayers = 10;
    unsigned int maxNumberOfLayerPairs = 13;
    unsigned int maxNumberOfRootLayerPairs = 3;

    constexpr const float theThetaCut = 0.002f;
    constexpr const float thePhiCut = 0.2f;
    constexpr const float theHardPtCut = 0.0f;

    // HOST ALLOCATIONS FOR THE INPUT
    //////////////////////////////////////
    GPUEvent *h_allEvents;
    unsigned int* h_indices;
    GPULayerDoublets* h_doublets;
    hipHostMalloc(&h_allEvents, nEvents * sizeof(GPUEvent));
    //per each event per each layerPair you have a max number of doublets
    gpuErrchk(
            hipHostMalloc(&h_indices,
                    nEvents * maxNumberOfLayerPairs * maxNumberOfDoublets * 2 * sizeof(int)));
    hipHostMalloc(&h_doublets, nEvents * maxNumberOfLayerPairs * sizeof(GPULayerDoublets));

    //per each event per each layer you have a max number of hits x y z
    GPULayerHits* h_layers;
    float *h_x, *h_y, *h_z;
    unsigned int* h_rootLayerPairs;

    hipHostMalloc(&h_layers, nEvents * maxNumberOfLayers * sizeof(GPULayerHits));
    gpuErrchk(hipHostMalloc(&h_x, nEvents * maxNumberOfLayers * maxNumberOfHits * sizeof(float)));
    hipHostMalloc(&h_y, nEvents * maxNumberOfLayers * maxNumberOfHits * sizeof(float));
    hipHostMalloc(&h_z, nEvents * maxNumberOfLayers * maxNumberOfHits * sizeof(float));
    hipHostMalloc(&h_rootLayerPairs, nEvents * maxNumberOfRootLayerPairs * sizeof(int));

    GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet> * h_foundNtuplets;
    hipHostMalloc(&h_foundNtuplets,
            eventsPreallocatedOnGPU * sizeof(GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet> ));

    for (unsigned int i = 0; i < nEvents; ++i)
    {
        h_allEvents[i].eventId = hostEvents[i].eventId;

        h_allEvents[i].numberOfRootLayerPairs = 0;

        h_allEvents[i].numberOfLayers = hostEvents[i].hitsLayers.size();
        h_allEvents[i].numberOfLayerPairs = hostEvents[i].doublets.size();
        for (unsigned int j = 0; j < maxNumberOfLayerPairs; ++j)
        {
            auto doubletIdx = i * maxNumberOfLayerPairs + j;
            h_doublets[doubletIdx].size = 0;
        }

        for (unsigned int j = 0; j < maxNumberOfLayers; ++j)
        {
            auto layerIdx = i * maxNumberOfLayers + j;

            h_layers[layerIdx].size = 0;

        }

        for (unsigned int j = 0; j < hostEvents[i].doublets.size(); ++j)
        {
            auto layerPairIndex = i * maxNumberOfLayerPairs + j;

            h_doublets[layerPairIndex].size = hostEvents[i].doublets[j].size;
            h_doublets[layerPairIndex].innerLayerId = hostEvents[i].doublets[j].innerLayerId;
            h_doublets[layerPairIndex].outerLayerId = hostEvents[i].doublets[j].outerLayerId;

            for (unsigned int l = 0; l < hostEvents[i].rootLayers.size(); ++l)
            {
                if (hostEvents[i].rootLayers[l] == h_doublets[layerPairIndex].innerLayerId)
                {
                    auto rootlayerPairId = i * maxNumberOfRootLayerPairs
                            + h_allEvents[i].numberOfRootLayerPairs;
                    h_rootLayerPairs[rootlayerPairId] = j;
                    h_allEvents[i].numberOfRootLayerPairs++;
                }

            }
            for (unsigned int l = 0; l < hostEvents[i].doublets[j].size; ++l)
            {
                auto hitId = layerPairIndex * maxNumberOfDoublets * 2 + 2 * l;
                h_indices[hitId] = hostEvents[i].doublets[j].indices[2 * l];
                h_indices[hitId + 1] = hostEvents[i].doublets[j].indices[2 * l + 1];

            }

        }

        for (unsigned int j = 0; j < hostEvents[i].hitsLayers.size(); ++j)
        {
            auto layerIdx = i * maxNumberOfLayers + j;

            h_layers[layerIdx].size = hostEvents[i].hitsLayers[j].size;
            h_layers[layerIdx].layerId = hostEvents[i].hitsLayers[j].layerId;
            for (unsigned int l = 0; l < hostEvents[i].hitsLayers[j].size; ++l)
            {
                auto hitId = layerIdx * maxNumberOfHits + l;

                h_x[hitId] = hostEvents[i].hitsLayers[j].x[l];
                h_y[hitId] = hostEvents[i].hitsLayers[j].y[l];
                h_z[hitId] = hostEvents[i].hitsLayers[j].z[l];

            }

        }

    }

#ifdef FP_DEBUG
    for (unsigned int i = 0; i < nEvents; ++i)
    {
        assert(h_allEvents[i].eventId == hostEvents[i].eventId);
        assert(h_allEvents[i].numberOfLayers == hostEvents[i].hitsLayers.size());
        assert(h_allEvents[i].numberOfLayerPairs == hostEvents[i].doublets.size());
        auto cellId=0;
        for (unsigned int j = 0; j < hostEvents[i].doublets.size();++j)
        {
            auto layerPairIdx = i*maxNumberOfLayerPairs+j;
            assert(h_doublets[layerPairIdx].size == hostEvents[i].doublets[j].size);

            for(unsigned int l = 0; l < hostEvents[i].doublets[j].size; ++l)
            {
                auto hitId = layerPairIdx*maxNumberOfDoublets*2 + 2*l;
                assert(hostEvents[i].doublets[j].indices[2*l] == h_indices[hitId]);
                assert(hostEvents[i].doublets[j].indices[2*l+1] == h_indices[hitId+1]);
                auto innerHitId = hostEvents[i].doublets[j].indices[2*l];
                auto outerHitId = hostEvents[i].doublets[j].indices[2*l+1];
                auto innerLayerId = hostEvents[i].doublets[j].innerLayerId;
                auto outerLayerId = hostEvents[i].doublets[j].outerLayerId;

                auto xinnerIndex = i*maxNumberOfLayers*maxNumberOfHits + maxNumberOfHits*innerLayerId + innerHitId;
                assert(hostEvents[i].hitsLayers[innerLayerId].x[innerHitId]==h_x[xinnerIndex] );
//                float x1, y1, z1, x2, y2, z2;
//
//                x1 = hostEvents[i].hitsLayers[innerLayerId].x[innerHitId];
//                y1 = hostEvents[i].hitsLayers[innerLayerId].y[innerHitId];
//                z1 = hostEvents[i].hitsLayers[innerLayerId].z[innerHitId];
//                x2 = hostEvents[i].hitsLayers[outerLayerId].x[outerHitId];
//                y2 = hostEvents[i].hitsLayers[outerLayerId].y[outerHitId];
//                z2 = hostEvents[i].hitsLayers[outerLayerId].z[outerHitId];
//                printf("\n\n\nCPU cellid %d innerhit outerhit (xyz) (%f %f %f), (%f %f %f)\n",cellId, x1,y1,z1,x2,y2,z2);

                cellId++;
            }

        }

    }
#endif

    int nGPUs;

    hipGetDeviceCount(&nGPUs);
    std::cout << "Number of available GPUs " << nGPUs << std::endl;
    std::cout << "Using " << numberOfCPUThreads << " CPU threads " << std::endl;

    omp_set_num_threads(numberOfCPUThreads);
    unsigned int numberOfCPUOnlyThreads = numberOfCPUThreads - nGPUs;
    // HOST WORKER ALLOCATIONS

    std::vector<std::vector<GPUCACell> > hostWorker_theCells;
    hostWorker_theCells.resize(numberOfCPUOnlyThreads);
    std::vector<std::vector<GPUSimpleVector<maxCellsPerHit, unsigned int> > > hostWorker_isOuterHitOfCell;
    hostWorker_isOuterHitOfCell.resize(numberOfCPUOnlyThreads);
    std::vector<GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet> > hostWorker_foundNtuplets;
    hostWorker_foundNtuplets.resize(numberOfCPUOnlyThreads);




    for(int i = 0; i <numberOfCPUOnlyThreads;++i )
    {
        hostWorker_theCells[i].resize(maxNumberOfLayerPairs * maxNumberOfDoublets);
        hostWorker_isOuterHitOfCell[i].resize(maxNumberOfLayers * maxNumberOfHits);



    }


    //GPU ALLOCATIONS
    std::cout << "preallocating memory on GPU " << std::endl;

    std::vector<Region*> d_regionParams;
    d_regionParams.resize(nGPUs);
    std::vector<GPUEvent*> d_events;
    d_events.resize(nGPUs);
    std::vector<unsigned int*> d_indices;
    d_indices.resize(nGPUs);
    std::vector<GPULayerDoublets*> d_doublets;
    d_doublets.resize(nGPUs);
    std::vector<GPULayerHits*> d_layers;
    d_layers.resize(nGPUs);
    std::vector<float*> d_x;
    d_x.resize(nGPUs);
    std::vector<float*> d_y;
    d_y.resize(nGPUs);
    std::vector<float*> d_z;
    d_z.resize(nGPUs);
    std::vector<unsigned int*> d_rootLayerPairs;
    d_rootLayerPairs.resize(nGPUs);
    std::vector < std::vector < hipStream_t >> streams;
    streams.resize(nGPUs);
    std::vector<GPUCACell*> device_theCells;
    device_theCells.resize(nGPUs);
    std::vector<GPUSimpleVector<maxCellsPerHit, unsigned int>*> device_isOuterHitOfCell;
    device_isOuterHitOfCell.resize(nGPUs);
    std::vector<GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet> *> d_foundNtuplets;
    d_foundNtuplets.resize(nGPUs);

    std::vector<std::vector<GPULayerHits*> > tmp_layers;
    tmp_layers.resize(nGPUs);
    std::vector<std::vector<GPULayerDoublets*> > tmp_layerDoublets;
    tmp_layerDoublets.resize(nGPUs);




    for (unsigned int gpuIndex = 0; gpuIndex < nGPUs; ++gpuIndex)
    {
        hipSetDevice(gpuIndex);

        tmp_layers[gpuIndex].resize(numberOfCUDAStreams);
        tmp_layerDoublets[gpuIndex].resize(numberOfCUDAStreams);

        hipMalloc(&d_regionParams[gpuIndex], sizeof(Region));
        hipMemcpy(d_regionParams[gpuIndex], h_regionParams, sizeof(Region),
                hipMemcpyHostToDevice);
        hipMalloc(&d_events[gpuIndex], eventsPreallocatedOnGPU * sizeof(GPUEvent));
        hipMalloc(&d_indices[gpuIndex],
                eventsPreallocatedOnGPU * maxNumberOfLayerPairs * maxNumberOfDoublets * 2
                        * sizeof(int));
        hipMalloc(&d_doublets[gpuIndex],
                eventsPreallocatedOnGPU * maxNumberOfLayerPairs * sizeof(GPULayerDoublets));
        hipMalloc(&d_layers[gpuIndex],
                eventsPreallocatedOnGPU * maxNumberOfLayers * sizeof(GPULayerHits));
        hipMalloc(&d_x[gpuIndex],
                eventsPreallocatedOnGPU * maxNumberOfLayers * maxNumberOfHits * sizeof(float));
        hipMalloc(&d_y[gpuIndex],
                eventsPreallocatedOnGPU * maxNumberOfLayers * maxNumberOfHits * sizeof(float));
        hipMalloc(&d_z[gpuIndex],
                eventsPreallocatedOnGPU * maxNumberOfLayers * maxNumberOfHits * sizeof(float));
        hipMalloc(&d_rootLayerPairs[gpuIndex],
                eventsPreallocatedOnGPU * maxNumberOfRootLayerPairs * sizeof(unsigned int));
        //////////////////////////////////////////////////////////
        // ALLOCATIONS FOR THE INTERMEDIATE RESULTS (STAYS ON WORKER)
        //////////////////////////////////////////////////////////

        hipMalloc(&device_theCells[gpuIndex],
                eventsPreallocatedOnGPU * maxNumberOfLayerPairs * maxNumberOfDoublets
                        * sizeof(GPUCACell));

        hipMalloc(&device_isOuterHitOfCell[gpuIndex],
                eventsPreallocatedOnGPU * maxNumberOfLayers * maxNumberOfHits
                        * sizeof(GPUSimpleVector<maxCellsPerHit, unsigned int> ));

        hipMemset(device_isOuterHitOfCell[gpuIndex], 0,
                eventsPreallocatedOnGPU * maxNumberOfLayers * maxNumberOfHits
                        * sizeof(GPUSimpleVector<maxCellsPerHit, unsigned int> ));
        //////////////////////////////////////////////////////////
        // ALLOCATIONS FOR THE RESULTS
        //////////////////////////////////////////////////////////

        hipMalloc(&d_foundNtuplets[gpuIndex],
                eventsPreallocatedOnGPU
                        * sizeof(GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet> ));



        streams[gpuIndex].resize(numberOfCUDAStreams);
        for (int i = 0; i < numberOfCUDAStreams; ++i)
        {

            hipHostMalloc(&tmp_layers[gpuIndex][i], maxNumberOfLayers * sizeof(GPULayerHits));
            hipHostMalloc(&tmp_layerDoublets[gpuIndex][i], maxNumberOfLayerPairs * sizeof(GPULayerDoublets));

            hipStreamCreate (&streams[gpuIndex][i]);
        }

    }

    //INITIALIZATION IS NOW OVER
    //HERE STARTS THE COMPUTATION

    tbb::concurrent_queue<unsigned int> queue;
    for (unsigned iteration = 0; iteration < numberOfIterations; iteration++)
    {
        for (unsigned int i = 0; i < nEvents; ++i)
        {
            queue.push(i);
        }
    }

    std::vector<tbb::concurrent_queue<unsigned int>> streamQueues(nGPUs);
    for (unsigned int gpuIndex = 0 ; gpuIndex < nGPUs ; ++gpuIndex)
    {
        for (unsigned int streamIndex = 0 ; streamIndex < numberOfCUDAStreams ; ++streamIndex)
        {
            streamQueues[gpuIndex].push(streamIndex);
        }
    }

    using tuple_t = std::tuple<tbb::concurrent_queue<unsigned int>*, unsigned int, unsigned int>;
    std::vector<std::vector<tuple_t>> backReferences(nGPUs);
    for (unsigned int gpuIndex = 0 ; gpuIndex < nGPUs ; ++gpuIndex)
    {
        backReferences[gpuIndex].resize(numberOfCUDAStreams);
        for (unsigned int streamIndex = 0 ; streamIndex < numberOfCUDAStreams ; ++streamIndex)
        {
            backReferences[gpuIndex][streamIndex] =
                std::make_tuple(&streamQueues[gpuIndex], gpuIndex, streamIndex);
        }
    }

#if defined(DEBUG)
    std::vector<std::vector<unsigned int>> nQuadruplets(nEvents);
    std::vector<std::mutex> mtx_nQuadruplets(nEvents);
    for (std::size_t n = 0 ; n < nEvents ; ++n) {
        nQuadruplets[n].reserve(numberOfIterations);
    }
    using dbgTup_t = std::tuple<std::vector<unsigned int>*, std::mutex*, GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet>*>;
    std::vector<std::vector<dbgTup_t>> dbgBackrefs(nGPUs, std::vector<dbgTup_t>(numberOfCUDAStreams));
#endif // defined(DEBUG)

std::vector<unsigned int> processedEventsPerThread;
processedEventsPerThread.resize(numberOfCPUThreads,0);
		std::cout << "Execution run will start in 3 second.\n" << std::endl;
		std::this_thread::sleep_for(std::chrono::seconds(3));
double start = omp_get_wtime();
#pragma omp parallel
    {
        unsigned int streamIndex = 0;
        int threadId = omp_get_thread_num();
        unsigned int gpuIndex;
        bool isGPUThread = false;
        if (threadId < nGPUs)
        {
            gpuIndex = threadId;
            isGPUThread = true;
        }


        if (isGPUThread)
        {
            while (!queue.empty())
            {

                hipSetDevice(gpuIndex);
                unsigned int i;
                queue.try_pop(i);
                while(!streamQueues[gpuIndex].try_pop(streamIndex));
                processedEventsPerThread[threadId]++;

                auto d_firstLayerPairInEvt = maxNumberOfLayerPairs * streamIndex;
                auto d_firstLayerInEvt = maxNumberOfLayers * streamIndex;
                auto d_firstDoubletInEvent = d_firstLayerPairInEvt * maxNumberOfDoublets;
                auto d_firstHitInEvent = d_firstLayerInEvt * maxNumberOfHits;

                auto h_firstLayerPairInEvt = maxNumberOfLayerPairs * i;
                auto h_firstLayerInEvt = maxNumberOfLayers * i;
                auto h_firstDoubletInEvent = h_firstLayerPairInEvt * maxNumberOfDoublets;
                auto h_firstHitInEvent = h_firstLayerInEvt * maxNumberOfHits;

                for (unsigned int j = 0; j < h_allEvents[i].numberOfLayerPairs; ++j)
                {
                    tmp_layerDoublets[gpuIndex][streamIndex][j] = h_doublets[h_firstLayerPairInEvt + j];
                    tmp_layerDoublets[gpuIndex][streamIndex][j].indices =
                            &d_indices[gpuIndex][d_firstDoubletInEvent * 2
                                    + j * maxNumberOfDoublets * 2];
                    hipMemcpyAsync(
                            &d_indices[gpuIndex][d_firstDoubletInEvent * 2
                                    + j * maxNumberOfDoublets * 2],
                            &h_indices[h_firstDoubletInEvent * 2 + j * maxNumberOfDoublets * 2],
                            tmp_layerDoublets[gpuIndex][streamIndex][j].size * 2 * sizeof(int),
                            hipMemcpyHostToDevice, streams[gpuIndex][streamIndex]);
                }

                for (unsigned int j = 0; j < h_allEvents[i].numberOfLayers; ++j)
                {
                    tmp_layers[gpuIndex][streamIndex][j]=h_layers[h_firstLayerInEvt + j];
                    tmp_layers[gpuIndex][streamIndex][j].x=&d_x[gpuIndex][d_firstHitInEvent
                                                                       + maxNumberOfHits * j];

                    hipMemcpyAsync(&d_x[gpuIndex][d_firstHitInEvent
                                                   + maxNumberOfHits * j],
                            &h_x[h_firstHitInEvent + j * maxNumberOfHits],
                            tmp_layers[gpuIndex][streamIndex][j].size * sizeof(float),
                            hipMemcpyHostToDevice, streams[gpuIndex][streamIndex]);


                    tmp_layers[gpuIndex][streamIndex][j].y = &d_y[gpuIndex][d_firstHitInEvent
                            + maxNumberOfHits * j];
                    hipMemcpyAsync(&d_y[gpuIndex][d_firstHitInEvent
                                                   + maxNumberOfHits * j],
                            &h_y[h_firstHitInEvent + j * maxNumberOfHits],
                            tmp_layers[gpuIndex][streamIndex][j].size * sizeof(float),
                            hipMemcpyHostToDevice, streams[gpuIndex][streamIndex]);


                    tmp_layers[gpuIndex][streamIndex][j].z = &d_z[gpuIndex][d_firstHitInEvent
                            + maxNumberOfHits * j];

                    hipMemcpyAsync(&d_z[gpuIndex][d_firstHitInEvent
                                                   + maxNumberOfHits * j],
                            &h_z[h_firstHitInEvent + j * maxNumberOfHits],
                            tmp_layers[gpuIndex][streamIndex][j].size * sizeof(float),
                            hipMemcpyHostToDevice, streams[gpuIndex][streamIndex]);
                }

                hipMemcpyAsync(
                        &d_rootLayerPairs[gpuIndex][maxNumberOfRootLayerPairs * streamIndex],
                        &h_rootLayerPairs[maxNumberOfRootLayerPairs * i],
                        h_allEvents[i].numberOfRootLayerPairs * sizeof(unsigned int),
                        hipMemcpyHostToDevice, streams[gpuIndex][streamIndex]);
                hipMemcpyAsync(&d_doublets[gpuIndex][d_firstLayerPairInEvt],
                        tmp_layerDoublets[gpuIndex][streamIndex],
                        h_allEvents[i].numberOfLayerPairs * sizeof(GPULayerDoublets),
                        hipMemcpyHostToDevice, streams[gpuIndex][streamIndex]);
                hipMemcpyAsync(&d_layers[gpuIndex][d_firstLayerInEvt],
                        tmp_layers[gpuIndex][streamIndex],
                        h_allEvents[i].numberOfLayers * sizeof(GPULayerHits),
                        hipMemcpyHostToDevice, streams[gpuIndex][streamIndex]);

                hipMemcpyAsync(&d_events[gpuIndex][streamIndex], &h_allEvents[i], sizeof(GPUEvent),
                        hipMemcpyHostToDevice, streams[gpuIndex][streamIndex]);

                dim3 numberOfBlocks_create(32, h_allEvents[i].numberOfLayerPairs);
                dim3 numberOfBlocks_connect(16, h_allEvents[i].numberOfLayerPairs);
                dim3 numberOfBlocks_find(8, h_allEvents[i].numberOfRootLayerPairs);
// KERNELS
//        debug_input_data<<<1,1,0,streams[streamIndex]>>>(&d_events[streamIndex], &d_doublets[d_firstLayerPairInEvt], &d_layers[d_firstLayerInEvt],d_regionParams,  maxNumberOfHits );
                kernel_create<<<numberOfBlocks_create,32,0,streams[gpuIndex][streamIndex]>>>(&d_events[gpuIndex][streamIndex], &d_doublets[gpuIndex][d_firstLayerPairInEvt],
                        &d_layers[gpuIndex][d_firstLayerInEvt], &device_theCells[gpuIndex][d_firstLayerPairInEvt*maxNumberOfDoublets],
                        &device_isOuterHitOfCell[gpuIndex][d_firstHitInEvent], &d_foundNtuplets[gpuIndex][streamIndex],d_regionParams[gpuIndex], maxNumberOfDoublets, maxNumberOfHits);

////
//        kernel_debug<<<1,1,0,streams[streamIndex]>>>(&d_events[streamIndex], &d_doublets[d_firstLayerPairInEvt],
//                &d_layers[d_firstLayerInEvt], &device_theCells[d_firstLayerPairInEvt*maxNumberOfDoublets],
//                &device_isOuterHitOfCell[d_firstHitInEvent], &d_foundNtuplets[streamIndex],
//                d_regionParams, theThetaCut, thePhiCut,theHardPtCut,maxNumberOfDoublets, maxNumberOfHits);
                kernel_connect<<<numberOfBlocks_connect,512,0,streams[gpuIndex][streamIndex]>>>(&d_events[gpuIndex][streamIndex],
                        &d_doublets[gpuIndex][d_firstLayerPairInEvt], &device_theCells[gpuIndex][d_firstLayerPairInEvt*maxNumberOfDoublets],
                        &device_isOuterHitOfCell[gpuIndex][d_firstHitInEvent], d_regionParams[gpuIndex], theThetaCut, thePhiCut,
                        theHardPtCut, maxNumberOfDoublets, maxNumberOfHits);

//        kernel_debug_connect<<<1,1,0,streams[streamIndex]>>>(&d_events[streamIndex], &d_doublets[d_firstLayerPairInEvt],
//                &device_theCells[d_firstLayerPairInEvt*maxNumberOfDoublets], &device_isOuterHitOfCell[d_firstHitInEvent],
//                 d_regionParams, maxNumberOfDoublets, maxNumberOfHits);
//        hipMemsetAsync(&d_foundNtuplets[streamIndex], 0, sizeof(GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet> ), streams[streamIndex]);

                kernel_find_ntuplets<<<numberOfBlocks_find,1024,0,streams[gpuIndex][streamIndex]>>>(&d_events[gpuIndex][streamIndex],
                        &d_doublets[gpuIndex][d_firstLayerPairInEvt], &device_theCells[gpuIndex][d_firstLayerPairInEvt*maxNumberOfDoublets],
                        &d_foundNtuplets[gpuIndex][streamIndex],&d_rootLayerPairs[gpuIndex][maxNumberOfRootLayerPairs*streamIndex], 4 , maxNumberOfDoublets);

//        kernel_debug_find_ntuplets<<<1,1,0,streams[streamIndex]>>>(&d_events[streamIndex],
//                &d_doublets[d_firstLayerPairInEvt], &device_theCells[d_firstLayerPairInEvt*maxNumberOfDoublets],
//                &d_foundNtuplets[streamIndex],&d_rootLayerPairs[maxNumberOfRootLayerPairs*streamIndex], 4 , maxNumberOfDoublets);
                hipMemcpyAsync(&h_foundNtuplets[streamIndex],
                        &d_foundNtuplets[gpuIndex][streamIndex],
                        sizeof(GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet> ),
                        hipMemcpyDeviceToHost, streams[gpuIndex][streamIndex]);
                hipMemsetAsync(&device_isOuterHitOfCell[gpuIndex][d_firstHitInEvent], 0,
                        maxNumberOfLayers * maxNumberOfHits
                                * sizeof(GPUSimpleVector<maxCellsPerHit, unsigned int> ),
                        streams[gpuIndex][streamIndex]);

//        hipStreamSynchronize(streams[streamIndex]);
//        std::cout << "found quadruplets " << h_foundNtuplets[streamIndex].size() << std::endl;

                // hipStreamSynchronize (streams[gpuIndex][streamIndex]);

#if defined(DEBUG)
                dbgBackrefs[gpuIndex][streamIndex] = dbgTup_t(&nQuadruplets[i], &mtx_nQuadruplets[i], &h_foundNtuplets[streamIndex]);
                hipStreamAddCallback(streams[gpuIndex][streamIndex],
                    [](hipStream_t, hipError_t, void *data) -> void
                    {
                        auto tup = static_cast<dbgTup_t*>(data);
                        auto vec = std::get<0>(*tup);
                        auto mtx = std::get<1>(*tup);
                        auto foundNtuplets = std::get<2>(*tup);
                        const auto sz = foundNtuplets->size();
                        std::lock_guard<std::mutex> guard(*mtx);
                        vec->push_back(sz);
                    },
                    static_cast<void*>(&dbgBackrefs[gpuIndex][streamIndex]),
                    0
                );
#endif // defined(DEBUG)

                hipStreamAddCallback(streams[gpuIndex][streamIndex],
                    [](hipStream_t, hipError_t, void *data) -> void
                    {
                        auto tup = static_cast<tuple_t*>(data);
                        auto queue = std::get<0>(*tup);
                        auto streamIndex = std::get<2>(*tup);
                        queue->push(streamIndex);
                    },
                    static_cast<void*>(&backReferences[gpuIndex][streamIndex]),
                    0
                );
            }

            for (int i = 0; i < numberOfCUDAStreams; ++i)
            {
                hipStreamSynchronize (streams[gpuIndex][i]);
            }

        }
        else
        {
            int CPUOnlyThreadId = threadId - nGPUs;

            while (!queue.empty())
            {
                unsigned int i;
                queue.try_pop(i);
                processedEventsPerThread[threadId]++;

                auto h_firstLayerPairInEvt = maxNumberOfLayerPairs * i;
                auto h_firstLayerInEvt = maxNumberOfLayers * i;
                auto h_firstDoubletInEvent = h_firstLayerPairInEvt * maxNumberOfDoublets;
                auto h_firstHitInEvent = h_firstLayerInEvt * maxNumberOfHits;
                std::vector<GPULayerDoublets> doublets;
                doublets.resize(h_allEvents[i].numberOfLayerPairs);


                for (unsigned int j = 0; j < h_allEvents[i].numberOfLayerPairs; ++j)
                {
                    doublets[j] = h_doublets[h_firstLayerPairInEvt + j];
                    doublets[j].indices =&h_indices[h_firstDoubletInEvent* 2 + j * maxNumberOfDoublets * 2];

                }

                std::vector<GPULayerHits> layerHits;
                layerHits.resize(h_allEvents[i].numberOfLayers);
                for (unsigned int j = 0; j < h_allEvents[i].numberOfLayers; ++j)
                {
                    layerHits[j] = h_layers[h_firstLayerInEvt + j];
                    layerHits[j].x=&h_x[h_firstHitInEvent + j * maxNumberOfHits];
                    layerHits[j].y=&h_y[h_firstHitInEvent + j * maxNumberOfHits];
                    layerHits[j].z=&h_z[h_firstHitInEvent + j * maxNumberOfHits];

                }

                host_kernel(&h_allEvents[i], doublets.data(),
                        layerHits.data(),
                        hostWorker_theCells[CPUOnlyThreadId],
                        hostWorker_isOuterHitOfCell[CPUOnlyThreadId], &h_rootLayerPairs[maxNumberOfRootLayerPairs * i], &hostWorker_foundNtuplets[CPUOnlyThreadId],
                        h_regionParams, theThetaCut, thePhiCut, theHardPtCut, maxNumberOfDoublets,
                        maxNumberOfHits);
                for(unsigned int j = 0; j < hostWorker_isOuterHitOfCell[CPUOnlyThreadId].size(); ++j)
                {
                    hostWorker_isOuterHitOfCell[CPUOnlyThreadId][j].reset();
                }

            }
        }

    }


double stop = omp_get_wtime();

#if defined(DEBUG) && defined(VERBOSE)
    for (std::size_t it = 0 ; it < numberOfIterations ; ++it) {
        std::cerr << "Iteration " << it << ":" << std::endl;
        for (std::size_t n = 0 ; n < nEvents ; ++n) {
            std::cerr << "    " << n << ": " << nQuadruplets[n][it] << std::endl;
        }
        std::cerr << std::endl;
    }
#endif // defined(DEBUG) && defined(VERBOSE)

#if defined(DEBUG) && defined(ASSERT)
    for (std::size_t n = 0 ; n < nEvents ; ++n) {
        assert(nQuadruplets[n].size() == numberOfIterations);
        assert(nQuadruplets[n].size() > 0);
        const auto ref = nQuadruplets[n][0];
        for (std::size_t it = 0 ; it < numberOfIterations ; ++it) {
            if (nQuadruplets[n][it] != ref) {
                std::cerr << "Event " << n << ", it. " << it << ": expected " << ref << ", got " << nQuadruplets[n][it] << " instead" << std::endl << std::flush;
            }
            assert(nQuadruplets[n][it] == ref);
        }
    }
#endif // defined(DEBUG) && defined(ASSERT)

    std::cout << "Summary: " << std::endl;
    unsigned int processedByGPU = 0;
    unsigned int processedByCPU = 0;

    for(unsigned int i = 0; i< processedEventsPerThread.size(); ++i)
    {
        std::cout << "\tthread " << i << " processed " << processedEventsPerThread[i] << " events." << std::endl;

        if(i < nGPUs)
            processedByGPU+=processedEventsPerThread[i];
        else
            processedByCPU+=processedEventsPerThread[i];
    }

    std::cout << numberOfIterations*nEvents << " events processed in " << stop-start << "s. Measured rate: " << numberOfIterations*nEvents/(stop-start) << " Hz " << std::endl;
    std::cout << processedByGPU << " events processed by " << nGPUs << " GPUs in  " << stop-start << "s. Measured GPU rate: " << processedByGPU/(stop-start) << " Hz " << std::endl;
    std::cout << processedByCPU << " events processed by " << numberOfCPUOnlyThreads << " CPUs in  " << stop-start << "s. Measured CPU rate: " << processedByCPU/(stop-start) << " Hz " << std::endl;

// CLEANUP


    std::cout << "deleting Device memory " << std::endl;

    for (unsigned int gpuIndex = 0; gpuIndex < nGPUs; ++gpuIndex)
    {
        hipSetDevice(gpuIndex);
        for (int i = 0; i < numberOfCUDAStreams; ++i)
        {
            hipStreamSynchronize (streams[gpuIndex][i]);

            hipStreamDestroy(streams[gpuIndex][i]);
            hipHostFree(tmp_layers[gpuIndex][i]);
            hipHostFree(tmp_layerDoublets[gpuIndex][i]);
        }

        hipFree(device_isOuterHitOfCell[gpuIndex]);
        hipFree(d_foundNtuplets[gpuIndex]);

        hipFree(d_regionParams[gpuIndex]);
        hipFree(device_theCells[gpuIndex]);

        hipFree(d_events[gpuIndex]);
        hipFree(d_indices[gpuIndex]);
        hipFree(d_doublets[gpuIndex]);
        hipFree(d_layers[gpuIndex]);
        hipFree(d_x[gpuIndex]);
        hipFree(d_y[gpuIndex]);
        hipFree(d_z[gpuIndex]);
        hipFree(d_rootLayerPairs[gpuIndex]);

    }
    std::cout << "deleting Host memory " << std::endl;

    hipHostFree(h_foundNtuplets);
    hipHostFree(h_regionParams);
    hipHostFree(h_allEvents);
    hipHostFree(h_layers);
    hipHostFree(h_x);
    hipHostFree(h_y);
    hipHostFree(h_z);
    hipHostFree(h_rootLayerPairs);
    hipHostFree(h_indices);
    hipHostFree(h_doublets);

    return 0;
}

